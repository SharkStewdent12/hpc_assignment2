#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "matrix.h"
#include "outputHandling.h"

const int imageDim=11;

int main( int argc, char** argv) {

	printError("test");
	//generate sample image
	Matrix sampleImage;
	sampleImage.width = imageDim;
	sampleImage.height = imageDim;


	populateRandomMatrix(sampleImage);

	printMatrix(sampleImage);


}



