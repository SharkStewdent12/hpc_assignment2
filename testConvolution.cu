///////////////////////////////////////////////////////////////////////////////
// Declarations
///////////////////////////////////////////////////////////////////////////////
// Libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Parameters
#define ImgDim 5
#define BlockSize 16
#define NumRepeats 10

// Functions (pre-declarations)
void applyConvolution_serial(int* sourceImg, int* kernel, int* resultImg);
__global__ void applyConvolution_parallel(int* sourceImg, int* kernel, int* resultImg);
bool equalsImage(int* image1, int* image2);
void displayMatrix(int* matrix, int matrixWidth, int matrixHeight);
void populateRandomImg(int* img);

///////////////////////////////////////////////////////////////////////////////
// Main funcion
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
	////////////////////
	// Kernel selection
	if (argc != 2){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	int kernelNum = strtol( argv[1], (char**)NULL,10 );

	/*
	if ((kernelNum < 1) || (kernelNum > 3)){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	*/

	printf("Kernel chosen: %d\n",kernelNum);

	////////////////////
	// Initialisation
	size_t imgSize = (ImgDim) * (ImgDim) * sizeof(int); //+2 to add border
	size_t imgSize_bordered = (ImgDim+2) * (ImgDim+2) * sizeof(int); //+2 to add border
	clock_t startTime;
	double serialTime, parallelTime;
	srand (time(NULL));
	int* sourceImg = (int*)malloc(imgSize_bordered);
	int* serialImg = (int*)malloc(imgSize);
	int* parallelImg = (int*)malloc(imgSize);
	int* kernel;// = (int*)malloc(9*sizeof(int));
	int kernel1[9] = {-1,-1,-1,-1,9,-1,-1,-1,-1};
	int kernel2[9] = {-1,0,1,-2,0,2,-1,0,1};
	int kernel3[9] = {2,0,0,0,-1,0,0,0,-1};

	switch (kernelNum) {
		case 1:
			kernel = kernel1;
			break;
		case 2:
			kernel = kernel2;
			break;
		case 3:
			kernel = kernel3;
			break;
		default:
			printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
			return 1;			
	}//end switch

	displayMatrix(kernel,3,3);//test code
	////////////////////
	//Generate image
	
	//Create border of zeros stars at a corner and works clockwise
	for (int pos = 0; pos < ImgDim+1; pos++) {
		sourceImg[pos] = 0; //top row, left to right
		sourceImg[(pos+1)*(ImgDim+2)-1] = 0; //right side, top to bottom
		sourceImg[(ImgDim+2)*(ImgDim+2)-pos-1] = 0; //bottom row, right to left
		sourceImg[(ImgDim+2-pos)*(ImgDim+2)] = 0; //left side, bottom to top

		serialImg[pos] = 0; //top row, left to right
		serialImg[(pos+1)*(ImgDim+2)-1] = 0; //right side, top to bottom
		serialImg[(ImgDim+2)*(ImgDim+2)-pos-1] = 0; //bottom row, right to left
		serialImg[(ImgDim+2-pos)*(ImgDim+2)] = 0; //left side, bottom to top

		parallelImg[pos] = 0; //top row, left to right
		parallelImg[(pos+1)*(ImgDim+2)-1] = 0; //right side, top to bottom
		parallelImg[(ImgDim+2)*(ImgDim+2)-pos-1] = 0; //bottom row, right to left
		parallelImg[(ImgDim+2-pos)*(ImgDim+2)] = 0; //left side, bottom to top

	}//end for pos

	for (int row = 1; row < ImgDim+1; row++) { //borders excluded
		for (int col = 1; col < ImgDim+1; col++) { //borders excluded
			sourceImg[row*(ImgDim+2) + col] = rand() % 266;
		}//end for col
	}//end for row

	displayMatrix(sourceImg, ImgDim + 2, ImgDim + 2);//test code

	////////////////////
	//time serial

	startTime = clock(); //start timer
	int sum;
	for (int repeat = 0; repeat < NumRepeats; repeat++) {
		//printf("serial repeat %i\n",repeat);

		for (int row = 1; row < ImgDim+1; row++) { //borders excluded
			for (int col = 1; col < ImgDim+1; col++) { //borders excluded
				
				sum = 0;

				//Convolution
				for (int rowOffset = -1; rowOffset <= 1; rowOffset++) {
					for (int colOffset = -1; colOffset <= 1; colOffset++) {
						
						sum = sum + (sourceImg[(row+rowOffset)*(ImgDim+2) + col + colOffset] * kernel[(rowOffset+1)*3 + (colOffset+1)] );

					}//end for colOffset
				}//end for rowOffset

				if (sum < 0) {
					sum = 0;
				}//end if
				serialImg[row*(ImgDim+2)+col] = sum;
				//serialImg[row*(ImgDim+2)+col] = 5;


			}//end for col
		}//end for row
		
	}//end for repeat

	displayMatrix(serialImg, ImgDim+2,ImgDim+2);

	serialTime = (double)(clock() - startTime) / CLOCKS_PER_SEC;

	////////////////////
	//time parallel
	startTime = clock(); //start timer
	for (int repeat = 0; repeat < NumRepeats; repeat++) {
		//printf("parallel repeat %i\n",repeat);
	}//end for repeat
	parallelTime = (double)(clock() - startTime) / CLOCKS_PER_SEC;

	////////////////////
	//validate output

	////////////////////
	//display results
	printf("%ix serial time: %f\n",NumRepeats,serialTime);
	printf("%ix parallel time: %f\n",NumRepeats,parallelTime);

	return 0; // That means it worked fine.
}

///////////////////////////////////////////////////////////////////////////////
// Functions
///////////////////////////////////////////////////////////////////////////////

void displayMatrix(int* matrix, int matrixWidth, int matrixHeight) {

for (int row = 0; row < matrixWidth; row++) {

	for (int col = 0; col < matrixHeight; col++) {

		printf("%i ",matrix[row*matrixWidth+col]);

	}//end for col
	
	printf("\n");

}//end for row


}//end funcion displayMatrix






















