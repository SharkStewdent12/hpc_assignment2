///////////////////////////////////////////////////////////////////////////////
// Declarations
///////////////////////////////////////////////////////////////////////////////
// Libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Parameters
#define ImgSize 256
#define BorderOffset 1
#define BlockSize 16
#define NumRepeats 10

// Functions (pre-declarations)
void applyConvolution_serial(int* sourceImg, int* kernel, int* resultImg);
__global__ void applyConvolution_parallel(int* sourceImg, int* kernel, int* resultImg);
bool equalsImage(int* image1, int* image2);
void displayImage(int* image);
void populateRandomImg(int* img);

///////////////////////////////////////////////////////////////////////////////
// Main funcion
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
	//initialise
	if (argc != 2){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	int kernelNum = strtol( argv[1], (char**)NULL,10 );
	if ((kernelNum < 1) || (kernelNum > 3)){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}

	printf("%d\n",kernelNum);

	//Generate image

	//choose kernel

	//time serial

	//time parallel

	//validate output

	//display results

	return 0;
}

///////////////////////////////////////////////////////////////////////////////
// Functions
///////////////////////////////////////////////////////////////////////////////
