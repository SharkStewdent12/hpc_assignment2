///////////////////////////////////////////////////////////////////////////////
// Declarations
///////////////////////////////////////////////////////////////////////////////
// Libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Parameters
#define ImgWidth 256
#define ImgHeight 256
#define BorderOffset 1
#define BlockSize 16
#define NumRepeats 10

// Functions (pre-declarations)
void applyConvolution_serial(int* sourceImg, int* kernel, int* resultImg);
__global__ void applyConvolution_parallel(int* sourceImg, int* kernel, int* resultImg);
bool equalsImage(int* image1, int* image2);
void displayMatrix(int* matrix, int matrixWidth, int matrixHeight);
void populateRandomImg(int* img);

///////////////////////////////////////////////////////////////////////////////
// Main funcion
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
	////////////////////
	// Kernel selection
	if (argc != 2){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	int kernelNum = strtol( argv[1], (char**)NULL,10 );

	/*
	if ((kernelNum < 1) || (kernelNum > 3)){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	*/

	printf("Kernel chosen: %d\n",kernelNum);

	////////////////////
	// Initialisation
	size_t imgSize = (ImgWidth+BorderOffset) * (ImgHeight+BorderOffset) * sizeof(int);
	srand (time(NULL));
	int* sourceImg = (int*)malloc(imgSize);
	int* serialImg = (int*)malloc(imgSize);
	int* parallelImg = (int*)malloc(imgSize);
	int* kernel;// = (int*)malloc(9*sizeof(int));
	int kernel1[9] = {-1,-1,-1,-1,9,-1,-1,-1,-1};
	int kernel2[9] = {-1,0,1,-2,0,2,-1,0,1};
	int kernel3[9] = {2,0,0,0,-1,0,0,0,-1};

	switch (kernelNum) {
		case 1:
			kernel = kernel1;
			break;
		case 2:
			kernel = kernel2;
			break;
		case 3:
			kernel = kernel3;
			break;
		default:
			printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
			return 1;			
	}//end switch

	displayMatrix(kernel,3,3);
	////////////////////
	//Generate image


	////////////////////
	//choose kernel

	////////////////////
	//time serial

	////////////////////
	//time parallel

	////////////////////
	//validate output

	////////////////////
	//display results

	return 0;
}

///////////////////////////////////////////////////////////////////////////////
// Functions
///////////////////////////////////////////////////////////////////////////////

void displayMatrix(int* matrix, int matrixWidth, int matrixHeight) {

for (int row = 0; row < matrixWidth; row++) {

	for (int col = 0; col < matrixHeight; col++) {

		printf("%d ",matrix[row*matrixWidth+col]);

	}//end for col
	
	printf("\n");

}//end for row


}//end funcion displayMatrix






















