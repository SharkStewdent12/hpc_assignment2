///////////////////////////////////////////////////////////////////////////////
// Declarations
///////////////////////////////////////////////////////////////////////////////
// Libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Parameters
#define ImgDim 5
#define BlockSize 16
#define NumRepeats 10

// Functions (pre-declarations)
void applyConvolution_serial(int* sourceImg, int* kernel, int* resultImg);
__global__ void applyConvolution_parallel(int* sourceImg, int* kernel, int* resultImg);
bool equalsImage(int* image1, int* image2);
void displayMatrix(int* matrix, int matrixWidth, int matrixHeight);
void populateRandomImg(int* img);

///////////////////////////////////////////////////////////////////////////////
// Main funcion
///////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) {
	////////////////////
	// Kernel selection
	if (argc != 2){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	int kernelNum = strtol( argv[1], (char**)NULL,10 );

	/*
	if ((kernelNum < 1) || (kernelNum > 3)){
		printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
		return 1;
	}
	*/

	printf("Kernel chosen: %d\n",kernelNum);

	////////////////////
	// Initialisation
	size_t imgSize = (ImgDim+1) * (ImgDim+1) * sizeof(int); //+1 to add border
	srand (time(NULL));
	int* sourceImg = (int*)malloc(imgSize);
	int* serialImg = (int*)malloc(imgSize);
	int* parallelImg = (int*)malloc(imgSize);
	int* kernel;// = (int*)malloc(9*sizeof(int));
	int kernel1[9] = {-1,-1,-1,-1,9,-1,-1,-1,-1};
	int kernel2[9] = {-1,0,1,-2,0,2,-1,0,1};
	int kernel3[9] = {2,0,0,0,-1,0,0,0,-1};

	switch (kernelNum) {
		case 1:
			kernel = kernel1;
			break;
		case 2:
			kernel = kernel2;
			break;
		case 3:
			kernel = kernel3;
			break;
		default:
			printf("Wrong arguments, please place a single number from 1 to 3 to choose a kernel to apply as an argument.\n");
			return 1;			
	}//end switch

	displayMatrix(kernel,3,3);
	////////////////////
	//Generate image
	
	//Create border of zeros stars at a corner and works clockwise
	for (int pos = 0; pos < ImgDim+1; pos++) {
		sourceImg[pos] = 0; //top row, left to right
		sourceImg[(pos+1)*(ImgDim+2)-1] = 0; //right side, top to bottom
		sourceImg[(ImgDim+2)*(ImgDim+2)-pos-1] = 0; //bottom row, right to left
		sourceImg[(ImgDim+2-pos)*(ImgDim+2)] = 0; //left side, bottom to top
	}//end for pos

	for (int row = 1; row < ImgDim+1; row++) { //borders excluded
		for (int col = 1; col < ImgDim+1; col++) { //borders excluded
			sourceImg[row*(ImgDim+2) + col] = rand() % 266;
		}//end for col
	}//end for row

	displayMatrix(sourceImg, ImgDim + 2, ImgDim + 2);

	////////////////////
	//choose kernel

	////////////////////
	//time serial

	////////////////////
	//time parallel

	////////////////////
	//validate output

	////////////////////
	//display results

	return 0; // That means it worked fine.
}

///////////////////////////////////////////////////////////////////////////////
// Functions
///////////////////////////////////////////////////////////////////////////////

void displayMatrix(int* matrix, int matrixWidth, int matrixHeight) {

for (int row = 0; row < matrixWidth; row++) {

	for (int col = 0; col < matrixHeight; col++) {

		printf("%i ",matrix[row*matrixWidth+col]);

	}//end for col
	
	printf("\n");

}//end for row


}//end funcion displayMatrix






















